#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <inttypes.h>
#include <string.h>
#include <iostream>
#include <chrono>
#include <ctime>

#include <setjmp.h>
//#include "libjpeg/jpeglib.h"

#define pi 3.14159265358979323846
#define itCount 100000
#define tol 0.01

using namespace std;
using namespace std::chrono;

float *arr, *d_arr;
unsigned short *buf, *h_buf;
float yOffset = 0;
float xOffset = 0;
int p = 0;

int red[3] = { 255, 255, 255 };
int black[3] = { 255, 0, 0 };

void init (unsigned short size) {
  __int64 buf_size = p*p*itCount*2*sizeof(unsigned short);
  __int64 img_size = size*size*sizeof(float);

  hipMalloc(&d_arr, img_size);
  hipMalloc(&buf, buf_size);
  
  double mem = (buf_size + img_size)/1024/1024;
  cout << buf_size/1024/1024 << " MB allocated for buf" << '\n';
  cout << img_size/1024/1024 << " MB allocated for img" << '\n';
  cout << mem << " MB allocated on GPU" << '\n';

  arr = (float*)malloc(img_size);
  h_buf = (unsigned short*)malloc(buf_size);

  for (int i = 0; i < size * size; i++) {
    arr[i] = 0;
  }
  for (int i = 0; i < p*p*itCount*2; i++) {
    h_buf[i] = 0;
  }
  hipMemcpy(d_arr, arr, img_size, hipMemcpyHostToDevice);
  hipMemcpy(buf, h_buf, buf_size, hipMemcpyHostToDevice);
}

__device__
float mod2Pi (float a) {
  double x = (double)a;
  double m = (double)(2 * pi);
  return fmod(fmod(x, m) + m, m);
}

__device__
int sround(float v, int i, unsigned short size) {
  if (v < 0) v += 2 * pi;
  if (v > 2 * pi) v -= 2 * pi;
  return round (v * (size - 1) / (2 * pi));
}

__device__
void Drw1 (float x, float y, float *arr, unsigned short* buf, unsigned short size, float K, int thr){
  int st = thr * 2 * itCount;
  float ep = 1;
  float eq = 0;
  float lsum = 0;
  for (int i = 0; i < itCount; i++) {
    x = mod2Pi (x + K * sin(y));
    y = mod2Pi (y + x);
    float dq = K * cos(y);
    float epn = ep + dq * eq;
    float eqn = ep + (1 + dq) * eq;
    ep = epn;
    eq = eqn;
    float dn = sqrt(ep * ep + eq * eq);
    ep = ep / dn;
    eq = eq / dn;
    lsum += log(dn);
    buf[st + i*2] = sround(mod2Pi(x), 0, size);
    buf[st + i*2 + 1] = sround(mod2Pi(y), 1, size);
  }
  float LLE = lsum / itCount;
  for (int i = 0; i < itCount; i++) {
    int ind = buf[st + i*2]*size + buf[st + i*2 + 1];
    arr[ind] = LLE;
  }
  __syncthreads();
}

__global__
void run (float step, float *arr, unsigned short *buf, unsigned short size, float K) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < 2 * pi / step) {
    float pp = i * step;
    Drw1 (pp, 0, arr, buf, size, K, i);
    Drw1 (0, pp, arr, buf, size, K, i);
  }
}

void show (char* fileName, unsigned short size) {
  FILE *f = fopen("testr.ppm", "wb");
  fprintf(f, "P6\n%i %i 255\n", size, size);
  float summ = 0;
  cout << "CHECK arr[0] " << arr[0] << '\n';
  for (int i = 0; i < size; i++)
    summ += arr[size * (size - 1) + i];
  cout << "CHECK1 " << summ << '\n';
  summ = 0;
  for (int i = 0; i < size; i++) {
    //if (arr[i] != 0) {
      //cout << fixed;
      //cout << i << ". " << arr[i] << '\n';
    //}
    summ += arr[i];
  }
  cout << "CHECK2 " << summ << '\n';
  // for (int i = 0; i < size * size; i++)
  //   summ += arr[i];
  // cout << "CHECK2 " << summ << '\n';
  for (int y = 0; y < size; y++) {
      for (int x = 0; x < size; x++) {
          int* color;
          if (arr[y * size + x] < tol) color = red;
          else color = black;
          fputc(color[0], f);
          fputc(color[1], f);
          fputc(color[2], f);
      }
  }
  fclose(f);
}

void show2 (char* fileName, unsigned short size) {
  float summ = 0;
  for (int y = 0; y < size; y++) {
    for (int x = 0; x < size; x++) {
      if (arr[y * size + x] >= 0) summ += arr[y * size + x];
      cout << arr[y * size + x] << " ";
    }
    cout << '\n';
  }
  cout << "summ: " << summ << '\n';
}

__int64 epoch() {
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

void std_map (float k, float step, unsigned short size, char *fileName) {
  char str[64];
  cout << "started\n" << "K = " << k << "\nstep = " << step << "\nsize = " << size << '\n';
  p = sqrt(2 * pi/step) + 1;
  init(size);
  printf("init finished\n");
  __int64 p1 = epoch();
  cout << p << " blocks" << '\n';
  cout << p*p << " threads" << '\n';
  run<<<p, p>>>(step, d_arr, buf, size, k);
  cout << "time taken: " << epoch() - p1 << '\n';
  hipMemcpy(arr, d_arr, size*size*sizeof(float), hipMemcpyDeviceToHost);
  cout << "time taken: " << epoch() - p1 << '\n';

  p1 = epoch();
  show(strcat(str, fileName), size);
  cout << "time taken: " << epoch() - p1 << '\n';

  free(arr);
  hipFree(d_arr);
  hipFree(buf);
}

float* parse_args (int c, char **args) {
  float* res = (float*)malloc(3*sizeof(float));
  res[0] = 1;
  res[1] = 0.1;
  res[2] = 1000.0;
  switch (c) {
    case 4:
      res[2] = atof (args[3]);
    case 3:
      res[1] = atof (args[2]);
    case 2:
      res[0] = atof (args[1]);
  }
  return res;
}

int main(int argc, char *argv[])
{
  char name1[64] = "qwe1132123.ppm";
  float* a = parse_args (argc, argv);
  std_map (a[0], a[1], (unsigned short)a[2], name1);
}