#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define _POSIX_C_SOURCE 199309L
#include <time.h>
#include <inttypes.h>
#include <string.h>
#include <iostream>
#include <chrono>
#include <ctime>
#define tPi 6.28318530718
#define itCount 100000
#define p 3

using namespace std;
using namespace std::chrono;

__global__
void simp(int *y, float step)
{
  int ind = blockIdx.x*blockDim.x + threadIdx.x;
  y[ind] = 2 * itCount * ind + 199999;
}


// void cpu_saxpy(float* y, float* x, int n) {
//   for (int i = 0; i < n; i++)
//     y[i] = a * x[i] + y[i];
// }

__int64 epoch() {
  return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

int main(void)
{
  int N = 200;
  printf("N: %i\n", N);
  int *d_y, *y;
  __int64 p1;
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    y[i] = 0;
  }

  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

  p1 = epoch();
  simp<<<1, N>>>(d_y, N);
  cout << "time taken: " << epoch() - p1 << '\n';
  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    cout << fixed << i << ". " << y[i] << '\n';
  }

  hipFree(d_y);
  free(y);
}